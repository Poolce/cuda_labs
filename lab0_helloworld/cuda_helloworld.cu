#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void taxpy_kernel(int arr_size, int* gpu_arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world. I am from %d block, %d thread (global index: %d)\n", blockIdx.x, threadIdx.x, i);
    if(i<arr_size) gpu_arr[i]+=i;
}

void cuda_helloworld(int arr_size,int* arr, int blocksPerGrid, int threadsPerBlock){
    hipError_t err = hipSuccess;

    //Allocation memory
    int* gpu_arr;
    err = hipMalloc((void**)&gpu_arr, arr_size*sizeof(int));
    if (err != hipSuccess){
        printf("gpuX memory allocation error. ");
        exit(EXIT_FAILURE);
    }

    //Relocation memory
    err = hipMemcpy(gpu_arr, arr, arr_size*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("Array memory relocation error. Host to device.\n%s",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Launch kernel
    taxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(arr_size, gpu_arr);
    hipDeviceSynchronize();

    //memory relocation Device to host
    err = hipMemcpy(arr, gpu_arr, arr_size*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        printf("Array memory relocation error. Device to host.\n%s",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //freeing memory 
    err = hipFree(gpu_arr);
    if (err != hipSuccess){
        printf("Array destruction error.\n%s",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(){
    int arr_size = 20;
    int* arr = new int[arr_size];
    for(int i = 0; i < arr_size; i++)
        arr[i] = i;

    cuda_helloworld(arr_size, arr, 5, 4);

    printf("[");
    for(int i = 0; i < arr_size; i++){
        if(i==(arr_size-1)){
            printf("%d]\n",arr[i]);
            break;
        } else {
            printf("%d, ",arr[i]);
        }
    }
}